
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

#define CSC(call)                                                   \
do {                                                                \
    hipError_t res = call;                                         \
    if (res != hipSuccess) {                                       \
        fprintf(stderr, "ERROR in %s:%d. Message: %s\n",            \
                __FILE__, __LINE__, hipGetErrorString(res));       \
        exit(0);                                                    \
    }                                                               \
} while(0)


__global__ void kernel(double *vector, int n) {
    int offset = blockDim.x * gridDim.x;
    
    for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < n; i += offset) {
        vector[i] *= vector[i] < 0 ? -1 : 1;
    }
}

int main() {
    int n;
    scanf("%d", &n);
    
    int size = n * sizeof(double);
    double *vector = (double *) malloc(size);
    for (int i = 0; i < n; ++i) {
        scanf("%lf", &vector[i]);
    }
    
    double *device_vector;

    CSC(hipMalloc(&device_vector, size));
    CSC(hipMemcpy(device_vector, vector, size, hipMemcpyHostToDevice));

    hipEvent_t start, end;
    CSC(hipEventCreate(&start));
    CSC(hipEventCreate(&end));
    CSC(hipEventRecord(start));

    kernel<<<1024, 1024>>>(device_vector, n);
    
    CSC(hipGetLastError());

    CSC(hipEventRecord(end));
    CSC(hipEventSynchronize(end));
    
    float time;
    CSC(hipEventElapsedTime(&time, start, end));
    CSC(hipEventDestroy(start));
    CSC(hipEventDestroy(end));

    printf("Time = %f ms\n", time);

    CSC(hipMemcpy(vector, device_vector, size, hipMemcpyDeviceToHost));
    CSC(hipFree(device_vector));

    /*
    for (int i = 0; i < n; ++i) {
        printf("%.10e ", vector[i]);
    }
    */

    printf("\n");
    free(vector);
    
    return 0;
}
